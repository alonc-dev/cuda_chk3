
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Kernel
__global__ void cuda_hello(){
    printf("Hello World from GPU (Device)!\n");
}

// Kernel definition
__global__ void VecAdd(int n, float* A, float* B, float* C)
{
    int i = threadIdx.x;
     if (i < n){
        C[i] = A[i] + B[i];
     }

}

__global__ void get_idx(int n, float x, float *a, float *b, float *c)
{

    int thIdx_x = threadIdx.x;
    int blIdx_x = blockIdx.x;
    int blDim_x = blockDim.x;

    printf("Running from GPU (Device) %d,%d,%d!\n",thIdx_x,blIdx_x,blDim_x);
    
    int i = thIdx_x*blIdx_x + blDim_x;
    if (i < n){
        a[i] = x;
        b[i] = (float)thIdx_x;
        c[i] = (float)thIdx_x;
    }

    a[0]=2.5;
    b[0]=2.5;
    c[0]=2.5;

}



// Main
int main() {

    printf("Start running on CPU (Host)!\n");

    int N = 1<<5;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // create x,y arr at host
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    c = (float*)malloc(N*sizeof(float));

    // create d_x, d_y arr at the device
    hipMalloc(&d_a, N*sizeof(float)); 
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
        c[i] = 3.0f;
    }

    // copy arr from host to device
    hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(N, d_a, d_a, d_c);
    // get_idx<<<(N+255)/256, 256>>>(N, 7.5f, d_a, d_b, d_c);

    // copy arr from device to host
    hipMemcpy(a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%2d/%d) \n", i, N);
        // printf("%2d/%d) a,b,c       = %f, %f, %f\n", i, N, a[i], b[i], c[i]);
        // printf("%2d/%d) d_a,d_b,d_c = %f, %f, %f\n", i, N, d_a[i], d_b[i], d_c[i]);
    }

    printf("End Running on CPU (Host)!\n");

    return 0;
}