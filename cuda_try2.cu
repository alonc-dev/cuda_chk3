
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Kernel
__global__ void cuda_hello(){
    printf("Hello World from GPU (Device)!\n");
}

// Kernel definition
__global__ void VecAdd(int n, float* A, float* B, float* C)
{

    int thIdx_x = threadIdx.x;

    printf("Running from GPU (Device %d)!\n",thIdx_x);

    int i = threadIdx.x;
     if (i < n){
        C[i] = A[i] + B[i] + (float)i;
     }

}


// Main
int main() {

    printf("Start running on CPU (Host)!\n");

    int N = 1<<5;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // create x,y arr at host
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    c = (float*)malloc(N*sizeof(float));

    // create d_x, d_y arr at the device
    hipMalloc(&d_a, N*sizeof(float)); 
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
        c[i] = 1.0f;
    }

    // copy arr from host to device
    hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(N, d_a, d_a, d_c);

    // copy arr from device to host
    hipMemcpy(a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        // printf("%2d/%d) \n", i, N);
        // printf("%2d/%d) d_a,d_b,d_c = %f, %f, %f\n", i, N, d_a[i], d_b[i], d_c[i]);
        printf("%2d/%d) a,b,c       = %f, %f, %f\n", i, N, a[i], b[i], c[i]);
    }

    printf("End Running on CPU (Host)!\n");

    return 0;
}