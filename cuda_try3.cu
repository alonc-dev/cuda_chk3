
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Kernel
__global__ void cuda_hello(){
    printf("Hello World from GPU (Device)!\n");
}

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

__global__ void get_idx(int n, float a, float *x, float *y)
{

    int thIdx_x = threadIdx.x;
    int blIdx_x = blockIdx.x;
    int blDim_x = blockDim.x;

    int thIdx_y = threadIdx.y;
    int blIdx_y = blockIdx.y;
    int blDim_y = blockDim.y;

    int thIdx_z = threadIdx.z;
    int blIdx_z = blockIdx.z;
    int blDim_z = blockDim.z;

    printf("Running from GPU (Device) %d,%d,%d!\n",thIdx_x,blIdx_x,blDim_x);
    
    int i = thIdx_x*blIdx_x + blDim_x;
    if (i < n){
        x[i] = a;
        y[i] = (float)thIdx_x;
    }

    x[0]=2.5;
    y[0]=2.5;
}



// Main
int main() {

    printf("Start running on CPU (Host)!\n");

    int N = 1<<5;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // create x,y arr at host
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    c = (float*)malloc(N*sizeof(float));

    // create d_x, d_y arr at the device
    hipMalloc(&d_a, N*sizeof(float)); 
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
        c[i] = 3.0f;
    }

    // copy arr from host to device
    hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(a, b, c);
    // get_idx<<<(N+255)/256, 256>>>(N, 7.5f, d_x, d_y);

    // copy arr from device to host
    hipMemcpy(a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%2d) a,b,c       = %f, %f, %f\n", i, a[i], b[i], c[i]);
        printf("%2d) d_a,d_b,d_c = %f, %f, %f\n", i, d_a[i], d_b[i], d_c[i]);
    }
    
    printf("End Running on CPU (Host)!\n");

    return 0;
}