#include <iostream>
#include <stdio.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

using namespace std;


// Kernel
__global__ 
void cuda_hello(){

    int idx = threadIdx.x;

    printf("identity: %d \n",idx);
    printf("Hello World from GPU! %d\n",idx);
    //cout << "Hello World cout GPU!" <<  "\n";
    //std::cout << "Hello World cout GPU!" << "\n";
}


// Main
int main() {
    printf("Hello World Start from CPU!\n");
    
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();

    printf("Hello World End from CPU!\n");
    return 0;
}