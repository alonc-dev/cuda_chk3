
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Kernel
__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
    
}

// Main
int main() {
    printf("Hello World from CPU!\n");
    
    cuda_hello<<<1,1>>>(); 
    return 0;
}